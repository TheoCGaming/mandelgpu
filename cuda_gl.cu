#include "hip/hip_runtime.h"
/*
 * This file is part of mandelgpu, a free GPU accelerated fractal viewer,
 * Copyright (C) 2016  Aksel Alpay
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "cuda_gl.hpp"
#include "cuda_error.hpp"
#include <GL/glew.h>
#include <GL/gl.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <cassert>

void cuda_gl::init_environment(int device_id)
{
  cudaGLSetGLDevice(device_id);
  check_cuda_error("cudaGLSetGLDevice failed!");
  glewInit();
}

cuda_gl::cuda_gl(const gl_renderer* r)
: _renderer(r)
{
  init();
}

cuda_gl::~cuda_gl()
{
  release();
}

void cuda_gl::rebuild_buffers()
{
  release();
  init();
}

void cuda_gl::release()
{
  hipGraphicsUnregisterResource(_resource);
  glDeleteTextures(1, &_texture);
  glDeleteBuffers(1, &_buffer);
}

void cuda_gl::init()
{
  glGenTextures( 1, &_texture );
  
  glBindTexture( GL_TEXTURE_2D, _texture );

  // set basic parameters
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
  
  glTexImage2D( GL_TEXTURE_2D, 0, GL_RGB, 
               _renderer->get_width(), _renderer->get_height(),
               0, GL_RGB, GL_UNSIGNED_BYTE, NULL );

  glBindTexture( GL_TEXTURE_2D, 0 );
  
  glGenBuffers( 1, &_buffer );
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER, _buffer );
  glBufferData( GL_PIXEL_UNPACK_BUFFER, 
               3 * _renderer->get_width() * _renderer->get_height(), 
               NULL, GL_STREAM_DRAW );
 
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER, 0 );
  

  hipGraphicsGLRegisterBuffer(&_resource, this->_buffer, 
                               hipGraphicsRegisterFlagsNone);
  check_cuda_error("hipGraphicsGLRegisterBuffer failed!");
  
  assert(glGetError() == GL_NO_ERROR);
}

void cuda_gl::display(std::function<void (unsigned char*, std::size_t, std::size_t)> kernel_call)
{
  glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT );
 
  hipGraphicsMapResources(1, &(this->_resource));
  check_cuda_error("hipGraphicsMapResources failed!");
  
  uchar3* pixels;
  std::size_t size;
  hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&pixels), 
                                       &size, _resource);
  check_cuda_error("hipGraphicsResourceGetMappedPointer failed!");
  
  // Call Kernel
  kernel_call(reinterpret_cast<unsigned char*>(pixels), _renderer->get_width(), _renderer->get_height());
  
  hipGraphicsUnmapResources(1, &(this->_resource));
  check_cuda_error("hipGraphicsUnmapResources failed!");
  
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, this->_buffer);
  glBindTexture(GL_TEXTURE_2D, this->_texture);
  
  glTexSubImage2D(GL_TEXTURE_2D, 0,
                  0, 0,
                  _renderer->get_width(), _renderer->get_height(),
                  GL_RGB, GL_UNSIGNED_BYTE, 0);
  
  glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
  
  //glBindTexture(GL_TEXTURE_2D, _texture);
  
  
  glEnable(GL_TEXTURE_2D);
  glDisable(GL_DEPTH_TEST);
  glDisable(GL_LIGHTING);
  glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

  glMatrixMode(GL_PROJECTION);
  glPushMatrix();
  glLoadIdentity();
  glOrtho(-1.0, 1.0, -1.0, 1.0, -1.0, 1.0);

  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();

  glViewport(0, 0, _renderer->get_width(), _renderer->get_height());


  glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 1.0f);
    glVertex2f(-1.0f, -1.0f);


    glTexCoord2f(1.0f, 1.0f);
    glVertex2f(1.0f, -1.0f);


    glTexCoord2f(1.0f, 0.0f);
    glVertex2f(1.0f, 1.0f);


    glTexCoord2f(0.0f, 0.0f);
    glVertex2f(-1.0f, 1.0f);
  glEnd();

  glMatrixMode(GL_PROJECTION);
  glPopMatrix();

  glDisable(GL_TEXTURE_2D);
  glGetError();
  //assert(glGetError() == GL_NO_ERROR);
}