#include "hip/hip_runtime.h"
/*
 * This file is part of mandelgpu, a free GPU accelerated fractal viewer,
 * Copyright (C) 2016  Aksel Alpay
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "kernel.hpp"
#include "cuda_error.hpp"

#define WITHOUT_HALF //half precision is not yet supported, as it would likely (?)
                     //require a different kernel
#ifndef WITHOUT_HALF
#include <hip/hip_fp16.h>
#endif


const int maxiterations = 2048;
const double limit = 1000.;

__device__ 
uchar3 hsv_to_rgb(const float h,
                  const float s,
                  const float v)
{
  int h_i = h / 60;
  
  float f = h / 60.f - h_i;
  
  float p = v * (1.f - s);
  float q = v * (1.f - s * f);
  float t = v * (1.f - s * (1.f - f));
  
  float3 rgb_temp;
  
  switch(h_i)
  {
  case 0:
    rgb_temp = make_float3(v,t,p);
    break;
  case 1:
    rgb_temp = make_float3(q,v,p);
    break;
  case 2:
    rgb_temp = make_float3(p,v,t);
    break;
  case 3:
    rgb_temp = make_float3(p,q,v);
    break;
  case 4:
    rgb_temp = make_float3(t,p,v);
    break;
  case 5:
    rgb_temp = make_float3(v,p,q);
    break;
  case 6:
    rgb_temp = make_float3(v,t,p);
    break;
  }
  
  float r = rgb_temp.x * 255.f;
  float g = rgb_temp.y * 255.f;
  float b = rgb_temp.z * 255.f;
  uchar3 result = make_uchar3((unsigned char)r,
                              (unsigned char)g,
                              (unsigned char)b);
  return result;
}

/*
template<unsigned n>
__device__ __forceinline__ arithmetic_type2 complex_power(const arithmetic_type2 z)
{
  arithmetic_type2 z_current = z;
  
  for(int i = 0; i < n - 1; ++i)
  {
    arithmetic_type2 old = z_current;
  }
  
  return z_current;
}

template<>
__device__ __forceinline__ arithmetic_type2 complex_power<1>(const arithmetic_type2 z)
{
  return z;
}
 * */

__device__
uchar3 color_scheme(int num_iterations, int max_iterations)
{
  uchar3 color;
  
  if(num_iterations == max_iterations)
    color = make_uchar3(0,0,0);
  else
  {
    float h = 360.f * (0.5f * sin(num_iterations * 2.f * M_PI / (float)max_iterations + 0.2f) + 0.5f);
    float s = 0.4f * sin(num_iterations * 0.5f * M_PI / 13.f) + 0.5f;
    float v = 0.3f * sin(num_iterations * 0.5f * M_PI / 200.f) + 0.7f;
    return hsv_to_rgb(h,s,v);
  }
  
  return color;
}

template<typename Arithmetic_type, typename Arithmetic_type2>
__global__
void mandelgpu(Arithmetic_type dx, Arithmetic_type center_x, Arithmetic_type center_y,
               uchar3* pixels, int npx_x, int npx_y)
{
  int gid_x = threadIdx.x + blockIdx.x * blockDim.x;
  int gid_y = threadIdx.y + blockIdx.y * blockDim.y;
  
  
  
  for(int px_x = gid_x; px_x < npx_x; px_x += blockDim.x * gridDim.x)
  {
    for(int px_y = gid_y; px_y < npx_y; px_y += blockDim.y * gridDim.y)
    {
      int iter_counter = 0;
      
      Arithmetic_type2 coord;
      coord.x = center_x + (px_x - npx_x / 2) * dx;
      coord.y = center_y + (px_y - npx_y / 2) * dx;
      
      Arithmetic_type2 z = coord;
      
#pragma unroll 128
      for(int i = 0; i < maxiterations; ++i)
      {         
        Arithmetic_type2 old_z = z;
        
        z = coord;
        
        Arithmetic_type a2 = 2.f * old_z.x;
        z.x += old_z.x * old_z.x;
        z.x -= old_z.y * old_z.y;
        z.y += a2 * old_z.y;
        
        Arithmetic_type norm2 = z.x * z.x;
        norm2 += z.y * z.y;

        if(norm2 > limit)
          break;
        ++iter_counter;
      }
      
      uchar3 color = color_scheme(iter_counter, maxiterations);
      pixels[px_y * npx_x + px_x] = color;
    }
  }
  
}

template<typename Arithmetic_type, typename Arithmetic_type2>
__global__
void juliagpu(Arithmetic_type dx, Arithmetic_type center_x, Arithmetic_type center_y,
               uchar3* pixels, int npx_x, int npx_y,
               const Arithmetic_type c0_x,
               const Arithmetic_type c0_y)
{
  int gid_x = threadIdx.x + blockIdx.x * blockDim.x;
  int gid_y = threadIdx.y + blockIdx.y * blockDim.y;
  
  
  for(int px_x = gid_x; px_x < npx_x; px_x += blockDim.x * gridDim.x)
  {
    for(int px_y = gid_y; px_y < npx_y; px_y += blockDim.y * gridDim.y)
    {
      int iter_counter = 0;
      
      Arithmetic_type2 coord;
      coord.x = center_x + (px_x - npx_x / 2) * dx;
      coord.y = center_y + (px_y - npx_y / 2) * dx;
      
      Arithmetic_type2 z = coord;
      
#pragma unroll 128
      for(int i = 0; i < maxiterations; ++i)
      { 
        Arithmetic_type2 old_z = z;
        
        z.x = c0_x;
        z.y = c0_y;

        Arithmetic_type a2 = 2.f * old_z.x;
        z.x += old_z.x * old_z.x;
        z.x -= old_z.y * old_z.y;
        z.y += a2 * old_z.y;
        
        Arithmetic_type norm2 = z.x * z.x;
        norm2 += z.y * z.y;

        if(norm2 > limit)
          break;
        
        ++iter_counter;
      }

      uchar3 color = color_scheme(iter_counter, maxiterations);
      pixels[px_y * npx_x + px_x] = color;
    }
  }
  
}

template<typename Arithmetic_type, typename Arithmetic_type2>
performance_estimator::result run_kernel(unsigned char* pixels, 
                std::size_t width, std::size_t height,
                double size_x, 
                double center_x, double center_y,
                double c0_x, double c0_y, // for julia
                kernel_type kernel)
{
  std::size_t npx_x = width;
  std::size_t npx_y = height;

  Arithmetic_type dx = size_x / static_cast<Arithmetic_type>(npx_x);
  
  std::size_t num_bytes = npx_x * npx_y * sizeof(uchar3);

  std::size_t nthreads = 8;
  std::size_t nblocks_x = npx_x / nthreads;
  std::size_t nblocks_y = npx_y / nthreads;
  
  if(npx_x % nthreads != 0)
    ++nblocks_x;
  if(npx_y % nthreads != 0)
    ++nblocks_y;
  
  dim3 threads = dim3(nthreads, nthreads, 1);
  dim3 blocks = dim3(nblocks_x, nblocks_y, 1);

  performance_estimator perf;
  perf.start();
  
  std::size_t num_bytes_transferred = num_bytes;
  std::size_t flops = npx_x * npx_y * (maxiterations * 10);
  switch(kernel)
  {
  case MANDELBROT:
    mandelgpu<Arithmetic_type, Arithmetic_type2><<<blocks, threads>>>(
                                   static_cast<Arithmetic_type>(dx), 
                                   static_cast<Arithmetic_type>(center_x), 
                                   static_cast<Arithmetic_type>(center_y), 
                                   reinterpret_cast<uchar3*>(pixels), 
                                   static_cast<int>(npx_x), 
                                   static_cast<int>(npx_y));
    break;
  case JULIA:
    juliagpu<Arithmetic_type, Arithmetic_type2><<<blocks, threads>>>(
                                  static_cast<Arithmetic_type>(dx), 
                                  static_cast<Arithmetic_type>(center_x), 
                                  static_cast<Arithmetic_type>(center_y), 
                                  reinterpret_cast<uchar3*>(pixels), 
                                  static_cast<int>(npx_x), 
                                  static_cast<int>(npx_y),
                                  c0_x, c0_y);

    break;
  }

  check_cuda_error("Kernel execution failed!");
  hipDeviceSynchronize();
  
  performance_estimator::result res = perf.stop(num_bytes_transferred, flops);
  
  check_cuda_error("Device synchronization failed!");
  
  return res;
}


performance_estimator::result run_kernel(unsigned char* pixels, 
                std::size_t width, std::size_t height,
                double size_x, 
                double center_x, double center_y,
                double c0_x, double c0_y, // for julia
                kernel_type kernel,
                precision p)
{
  switch(p)
  {
#ifndef WITHOUT_HALF
  case HALF:
    return run_kernel<half, half2>(pixels, 
                            width, height, 
                            size_x, center_x, center_y, 
                            c0_x, c0_y, kernel);
#endif
  case SINGLE:
    return run_kernel<float, float2>(pixels, 
                            width, height, 
                            size_x, center_x, center_y, 
                            c0_x, c0_y, kernel);
  case DOUBLE:
    return run_kernel<double, double2>(pixels, 
                            width, height, 
                            size_x, center_x, center_y, 
                            c0_x, c0_y, kernel);
  default:
    return run_kernel<double, double2>(pixels, 
                            width, height, 
                            size_x, center_x, center_y, 
                            c0_x, c0_y, kernel);
  }
  
}
